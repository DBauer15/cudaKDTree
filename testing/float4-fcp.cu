#include "hip/hip_runtime.h"
#include "cukd/builder.h"
// fcp = "find closest point" query
#include "cukd/fcp.h"

float4 *generatePoints(int N)
{
  std::cout << "generating " << N <<  " points" << std::endl;
  float4 *d_points = 0;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_points,N*sizeof(float4)));
  for (int i=0;i<N;i++) {
    d_points[i].x = (float)drand48();
    d_points[i].y = (float)drand48();
    d_points[i].z = (float)drand48();
    d_points[i].w = (float)drand48();
  }
  return d_points;
}

__global__ void d_fcp(int *d_results,
                    float4 *d_queries,
                    int numQueries,
                    float4 *d_nodes,
                    int numNodes)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;

  d_results[tid] = cukd::fcp(d_queries[tid],d_nodes,numNodes);
}

void fcp(int *d_results,
         float4 *d_queries,
         int numQueries,
         float4 *d_nodes,
         int numNodes)
{
  int bs = 128;
  int nb = cukd::common::divRoundUp(numQueries,bs);
  d_fcp<<<nb,bs>>>(d_results,d_queries,numQueries,d_nodes,numNodes);
}

bool noneBelow(float4 *d_points, int N, int curr, int dim, float value)
{
  if (curr >= N) return true;
  return
    ((&d_points[curr].x)[dim] >= value)
    && noneBelow(d_points,N,2*curr+1,dim,value)
    && noneBelow(d_points,N,2*curr+2,dim,value);
}

bool noneAbove(float4 *d_points, int N, int curr, int dim, float value)
{
  if (curr >= N) return true;
  return
    ((&d_points[curr].x)[dim] <= value)
    && noneAbove(d_points,N,2*curr+1,dim,value)
    && noneAbove(d_points,N,2*curr+2,dim,value);
}

bool checkTree(float4 *d_points, int N, int curr=0)
{
  if (curr >= N) return true;

  int dim = cukd::BinaryTree::levelOf(curr)%4;
  float value = (&d_points[curr].x)[dim];
  
  if (!noneAbove(d_points,N,2*curr+1,dim,value))
    return false;
  if (!noneBelow(d_points,N,2*curr+2,dim,value))
    return false;
  
  return
    checkTree(d_points,N,2*curr+1)
    &&
    checkTree(d_points,N,2*curr+2);
}

int main(int ac, const char **av)
{
  using namespace cukd::common;
  
  int N = 173;
  if (ac > 1) N = std::stoi(av[1]);
  
  float4 *d_points = generatePoints(N);

  {
    double t0 = getCurrentTime();
    std::cout << "calling builder..." << std::endl;
    cukd::buildTree<float4,float>(d_points,N);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took " << prettyDouble(t1-t0) << "s" << std::endl;
  }

  std::cout << "checking tree..." << std::endl;
  if (!checkTree(d_points,N))
    throw std::runtime_error("not a valid kd-tree!?");
  else
    std::cout << "... passed" << std::endl;

  float4 *d_queries = generatePoints(N);
  int    *d_results;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_results,N*sizeof(int)));
  {
    double t0 = getCurrentTime();
    fcp(d_results,d_queries,N,d_points,N);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done fcp query, took " << prettyDouble(t1-t0) << "s" << std::endl;
  }

  std::cout << "verifying ..." << std::endl;
  for (int i=0;i<N;i++) {
    if (d_results[i] == -1) continue;
    
    float4 qp = d_queries[i];
    float reportedDist = cukd::distance(qp,d_points[d_results[i]]);
    for (int j=0;j<N;j++) {
      float dist_j = cukd::distance(qp,d_points[j]);
      if (dist_j < reportedDist) {
        printf("for query %i: found offending point %i (%f %f %f %f) with dist %f (vs %f)\n",
               i,
               j,
               d_points[j].x,
               d_points[j].y,
               d_points[j].z,
               d_points[j].w,
               dist_j,
               reportedDist);
               
        throw std::runtime_error("verification failed ...");
      }
    }
  }
  std::cout << "verification succeeded... done." << std::endl;
}
